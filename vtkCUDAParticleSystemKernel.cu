#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * CUDA particle system kernel code.
 */

#include <stdio.h>
#include <math.h>
#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "vtkCUDAParticleSystemKernel.cuh"

#if USE_TEX
// textures for particle position and velocity
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;

texture<uint, 1, hipReadModeElementType> gridParticleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<uint, 1, hipReadModeElementType> cellEndTex;
#endif

// simulation parameters in constant memory
__constant__ SimParams params;


struct integrate_functor
{
	float deltaTime;

	__host__ __device__
	integrate_functor(float delta_time) : deltaTime(delta_time) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		volatile float4 posData = thrust::get<0>(t);
		volatile float4 velData = thrust::get<1>(t);
		float3 pos = make_float3(posData.x, posData.y, posData.z);
		float3 vel = make_float3(velData.x, velData.y, velData.z);

		//vel += params.gravity * deltaTime;
		//vel *= params.globalDamping;

		// new position = old position + velocity * deltaTime
		pos += vel * deltaTime;

		//if (pos.y < -1.0f + params.particleRadius) { pos.y = -1.0f + params.particleRadius; vel.y *= params.boundaryDamping;}

		// store new position and velocity
		thrust::get<0>(t) = make_float4(pos, posData.w);
		thrust::get<1>(t) = make_float4(vel, velData.w);
	}
};
