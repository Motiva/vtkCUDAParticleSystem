#include "hip/hip_runtime.h"
#include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "vtkCUDAParticleSystemKernel.cu"

extern "C"
{

void cudaInit()
//void cudaInit(int argc, char **argv)
{
	int devID = cutGetMaxGflopsDeviceId();
	hipSetDevice( devID );
}

void cudaGLInit(int argc, char **argv)
{
	cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
}

void allocateArray(void **devPtr, size_t size)
{
	cutilSafeCall(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
	cutilSafeCall(hipFree(devPtr));
}

void threadSync()
{
	cutilSafeCall(cutilDeviceSynchronize());
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
	cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void copyArrayFromDevice(void* host, const void* device,
		struct hipGraphicsResource **cuda_vbo_resource, int size)
{
	//if (cuda_vbo_resource)
	//	device = mapGLBufferObject(cuda_vbo_resource);

	cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

	//if (cuda_vbo_resource)
	//	unmapGLBufferObject(*cuda_vbo_resource);
}

/*void setParameters(SimParams *hostParams)
{
	// copy parameters to constant memory
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}*/

void integrateSystem(float *pos,
		float *vel,
		float *acc,
		float deltaTime,
		uint numParticles)
{
	thrust::device_ptr<float4> d_pos4((float4 *)pos);
	thrust::device_ptr<float4> d_vel4((float4 *)vel);
	thrust::device_ptr<float4> d_acc4((float4 *)acc);

	thrust::for_each(
				thrust::make_zip_iterator(thrust::make_tuple(d_vel4, d_acc4)),
				thrust::make_zip_iterator(thrust::make_tuple(d_vel4+numParticles, d_acc4+numParticles)),
				integrate_functor(deltaTime));
	thrust::for_each(
			thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
			thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
			integrate_functor(deltaTime));
}

/*void calcHash(uint*  gridParticleHash,
		uint*  gridParticleIndex,
		float* pos,
		int    numParticles)
{
	uint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);

	// execute the kernel
	calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
			gridParticleIndex,
			(float4 *) pos,
			numParticles);

	// check if kernel invocation generated an error
	cutilCheckMsg("Kernel execution failed");
}*/


void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
	thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
			thrust::device_ptr<uint>(dGridParticleHash + numParticles),
			thrust::device_ptr<uint>(dGridParticleIndex));
}

}   // extern "C"
